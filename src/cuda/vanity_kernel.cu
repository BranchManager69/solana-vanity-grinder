

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Seeds for random number generation
extern "C" __global__ void init_rng(hiprandState *states, unsigned long long seed, int num_keys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_keys) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// Base58 character set used by Solana
__device__ const char BASE58_CHARS[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

// Checks if the base58 encoded address matches the given pattern
__device__ bool check_pattern(const char* encoded, const char* pattern, int pattern_len, 
                              bool is_prefix, bool case_sensitive) {
    if (is_prefix) {
        for (int i = 0; i < pattern_len; i++) {
            char c1 = encoded[i];
            char c2 = pattern[i];
            
            if (!case_sensitive) {
                // Simple lowercase conversion for ASCII
                if (c1 >= 'A' && c1 <= 'Z') c1 += 32;
                if (c2 >= 'A' && c2 <= 'Z') c2 += 32;
            }
            
            if (c1 != c2) return false;
        }
        return true;
    } else {
        // Suffix matching
        int addr_len = 0;
        while (encoded[addr_len] != '\0') addr_len++;
        
        if (addr_len < pattern_len) return false;
        
        for (int i = 0; i < pattern_len; i++) {
            char c1 = encoded[addr_len - pattern_len + i];
            char c2 = pattern[i];
            
            if (!case_sensitive) {
                // Simple lowercase conversion for ASCII
                if (c1 >= 'A' && c1 <= 'Z') c1 += 32;
                if (c2 >= 'A' && c2 <= 'Z') c2 += 32;
            }
            
            if (c1 != c2) return false;
        }
        return true;
    }
}

// Simple Base58Check encoding - simplified for pattern matching only
__device__ void encode_base58_check(const unsigned char* data, int len, char* output) {
    // This is a simplified version that doesn't do proper Base58Check
    // but is sufficient for pattern matching
    
    int out_idx = 0;
    
    // Just encode the first few bytes for pattern matching
    for (int i = 0; i < min(len, 8); i++) {
        output[out_idx++] = BASE58_CHARS[data[i] % 58];
    }
    
    output[out_idx] = '\0';
}

// Main kernel for generating and checking keypairs
extern "C" __global__ void generate_and_check_keypairs(
    hiprandState *states,
    unsigned char *seed_data,
    unsigned char *result_keypair,
    int *found_flag,
    char *pattern,
    int pattern_len,
    bool is_prefix,
    bool case_sensitive,
    int num_keys
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_keys || *found_flag > 0) return;
    
    // Generate random seed data (32 bytes for ed25519)
    unsigned char seed[32];
    int offset = idx * 32;
    
    hiprandState localState = states[idx];
    for (int i = 0; i < 32; i++) {
        seed[i] = (unsigned char)(hiprand(&localState) % 256);
    }
    
    // Copy seed to global memory for later processing
    for (int i = 0; i < 32; i++) {
        seed_data[offset + i] = seed[i];
    }
    
    // Simplified: we'll use the seed as the public key for now
    // In the actual implementation, this would be replaced with proper ed25519 derivation
    char encoded_address[64];
    encode_base58_check(seed, 32, encoded_address);
    
    // Check if the address matches the pattern
    if (check_pattern(encoded_address, pattern, pattern_len, is_prefix, case_sensitive)) {
        // If we found a match and no one else has, copy the keypair to the result
        if (atomicExch(found_flag, 1) == 0) {
            for (int i = 0; i < 32; i++) {
                result_keypair[i] = seed[i];
            }
        }
    }
}

// This is the old function kept for backward compatibility with the benchmarking code
extern "C" __global__ void generate_keypairs(unsigned char *keys, int num_keys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_keys) {
        int offset = idx * 32;
        for (int i = 0; i < 32; i++) {
            keys[offset + i] = (unsigned char)((idx + i) % 256);
        }
    }
}
        